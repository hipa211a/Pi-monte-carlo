#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <memory>

constexpr int BLOCKS = 256;
constexpr int THREADS_PER_BLOCK = 256;

__global__ void monte_carlo_pi(unsigned long long* count, unsigned long long iterations, hiprandState* states) {
    unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned long long local_count = 0;
    hiprandState localState = states[tid];

    for (unsigned long long i = 0; i < iterations; i++) {
        float x = hiprand_uniform(&localState);
        float y = hiprand_uniform(&localState);
        if (x*x + y*y <= 1.0f) {
            local_count++;
        }
    }

    atomicAdd(count, local_count);
}

__global__ void setup_random_states(hiprandState* states) {
    unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(clock64(), tid, 0, &states[tid]);
}

struct CudaDeleter {
    void operator()(void* ptr) const {
        hipFree(ptr);
    }
};

template<typename T>
using CudaUniquePtr = std::unique_ptr<T, CudaDeleter>;

template<typename T>
CudaUniquePtr<T> make_cuda_unique(size_t size) {
    T* ptr;
    hipMalloc(&ptr, size);
    return CudaUniquePtr<T>(ptr);
}

int main() {
    unsigned long long h_count = 0;
    constexpr unsigned long long iterations = 1000000;

    auto d_count = make_cuda_unique<unsigned long long>(sizeof(unsigned long long));
    auto d_states = make_cuda_unique<hiprandState>(BLOCKS * THREADS_PER_BLOCK * sizeof(hiprandState));

    hipMemcpy(d_count.get(), &h_count, sizeof(unsigned long long), hipMemcpyHostToDevice);

    setup_random_states<<<BLOCKS, THREADS_PER_BLOCK>>>(d_states.get());

    monte_carlo_pi<<<BLOCKS, THREADS_PER_BLOCK>>>(d_count.get(), iterations, d_states.get());

    hipMemcpy(&h_count, d_count.get(), sizeof(unsigned long long), hipMemcpyDeviceToHost);

    double pi_estimate = 4.0 * static_cast<double>(h_count) / static_cast<double>(iterations * BLOCKS * THREADS_PER_BLOCK);

    std::cout << "Estimated Pi: " << pi_estimate << std::endl;
    std::cout << "Difference from M_PI: " << std::abs(pi_estimate - M_PI) << std::endl;

    return 0;
}